#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <CL/cl.h>
#include <time.h>

//OpenCL kernel function string to be fed in each device's work-item				(V.IMP)
const char *saxpy_kernel = "__kernel void saxpy_kernel(float alpha, 	"
		     "				__global float *A,	"
		     "				__global float *B,	"
		     "				__global float *C){	"
		     "		//Get global index of work item		\n"
		     "		int id = get_global_id(0);		"
		     "		C[id] = alpha * A[id] + B[id];		"
		     "	    }						";

int main(int argc, char **argv){
	if (argc != 3){
		printf("Usage : ./prog7 <vector size> <local size/threads>\n");
		exit(1);
	}

	unsigned int VECTOR_SIZE = atoi(argv[1]);
	int i;
	double start,end;

	// Allocate A,B,C in host memory
	float *A = (float *)calloc(VECTOR_SIZE,sizeof(float));
	float *B = (float *)calloc(VECTOR_SIZE,sizeof(float));
	float *C = (float *)calloc(VECTOR_SIZE,sizeof(float));
	float alpha = 2.0;
	for (i = 0; i < VECTOR_SIZE; i++){
		A[i] = i;
		B[i] = VECTOR_SIZE - i;
	}

	cl_int clstat;
	start = clock();
	
	// Get platform and device info (ids n nums)
	cl_platform_id *platforms = NULL;
	cl_uint num_platforms;
	clGetPlatformIDs(0,NULL,&num_platforms);
	platforms = (cl_platform_id *)malloc(num_platforms * sizeof(cl_platform_id));    			
	clGetPlatformIDs(1,platforms,NULL);

	cl_device_id *device_list = NULL;
	cl_uint num_devices;
	clGetDeviceIDs(platforms[0],CL_DEVICE_TYPE_GPU,0,NULL,&num_devices);
	device_list = (cl_device_id *)calloc(num_devices,sizeof(cl_device_id));
	clGetDeviceIDs(platforms[0],CL_DEVICE_TYPE_GPU,num_devices,device_list,NULL);

	// Create OpenCL context for each device
	cl_context context = clCreateContext(NULL,num_devices,device_list,NULL, NULL, &clstat);

	// Create Command Queue
	cl_command_queue cmd_queue = clCreateCommandQueueWithProperties(context,device_list[0],NULL,&clstat);
	
	// Create Memory Buffers (buffer objects)
	cl_mem A_clmem = clCreateBuffer(context,CL_MEM_READ_ONLY,VECTOR_SIZE * sizeof(float),NULL,&clstat);		//vimp - (context,flag,size,NULL,NULL)
	cl_mem B_clmem = clCreateBuffer(context,CL_MEM_READ_ONLY,VECTOR_SIZE * sizeof(float),NULL,&clstat);
	cl_mem C_clmem = clCreateBuffer(context,CL_MEM_WRITE_ONLY,VECTOR_SIZE * sizeof(float),NULL,&clstat);

	// Copy host objects into device memory
	clEnqueueWriteBuffer(cmd_queue,A_clmem,CL_TRUE,0,VECTOR_SIZE * sizeof(float),A,0,NULL,NULL);		//vimp - (cq, cl_mem, block_write, offset, size, host_mem, 0, NULL, NULL)
	clEnqueueWriteBuffer(cmd_queue,B_clmem,CL_TRUE,0,VECTOR_SIZE * sizeof(float),B,0,NULL,NULL);

	// Create Program From Kernel Source
	cl_program prg = clCreateProgramWithSource(context,1,(const char **)&saxpy_kernel, NULL, &clstat);	//vimp - (context, count, **kernel_string, lengths = NULL, NULL)

	// Build Program Object
	clBuildProgram(prg,num_devices,device_list,0,NULL,NULL);						//vimp - (programobj, num_devices, device_list[0], events = 0,NULL,NULL)

	// Create OpenCL kernel for each device
	cl_kernel kernel = clCreateKernel(prg, "saxpy_kernel", NULL);						//vimp - (programobj, func_name. NULL)

	// Set Kernel Args
	clSetKernelArg(kernel,0,sizeof(float),(void *)&alpha);						//vimp - (kernel, arg_pos, sizeof, arg_var)
	clSetKernelArg(kernel,1,sizeof(cl_mem),(void *)&A_clmem);						
	clSetKernelArg(kernel,2,sizeof(cl_mem),(void *)&B_clmem);						
	clSetKernelArg(kernel,3,sizeof(cl_mem),(void *)&C_clmem);						
	
	// EXECUTE KERNEL FUNCTION
	size_t global_size = VECTOR_SIZE;		//Process all vector items
	size_t local_size = (size_t)atoi(argv[2]); 	//Set no. of work-items in workgrp
	
	clEnqueueNDRangeKernel(cmd_queue,kernel,1,0,&global_size,&local_size,0,NULL,NULL);		//vimp - (cq, kernel, dim, offset, *glob_s, *local_s, events = 0, NULL, NULL)

	// Copy results back to host mem
	clEnqueueReadBuffer(cmd_queue,C_clmem,CL_TRUE,0,VECTOR_SIZE * sizeof(float),C,0,NULL,NULL);

	// Flush and clean command queue
	clFlush(cmd_queue);
	clFinish(cmd_queue);

	end = clock();

	// Print results
	//for (i = 0; i < VECTOR_SIZE; i++){
	//	printf("%f * %f + %f = %f\n",alpha,A[i],B[i],C[i]);
	//}
	printf("\n\nTime taken = %lf sec\n",(end - start)/CLOCKS_PER_SEC);

	// RELEASE
	clReleaseKernel(kernel);
	clReleaseProgram(prg);
	clReleaseMemObject(A_clmem);
	clReleaseMemObject(B_clmem);
	clReleaseMemObject(C_clmem);
	clReleaseCommandQueue(cmd_queue);
	clReleaseContext(context);
	free(A);
	free(B);
	free(C);
	return 0;
}
